#include "hip/hip_runtime.h"
/**
\example cuda-interop.cu

Cuda interoperability example program
*/

#include <assert.h>
#include <goopax>
#include <iostream>
#include <vector>

using std::cerr;
using std::cout;
using std::endl;

template<typename T>
std::ostream& operator<<(std::ostream& s, const std::vector<T>& v)
{
    s << "(";
    for (int k = 0; k < v.size(); ++k)
    {
        if (k != 0)
            s << ",";
        s << v[k];
    }
    s << ")";
    return s;
}

void check_cuda(hipError_t err)
{
    if (err != hipSuccess)
    {
        cerr << "CUDA error: " << hipGetErrorString(err) << endl;
        abort();
    }
}

__global__ void inc_cuda(float* C, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {
        C[i] += 1;
    }
}

int main()
{
    std::vector<float> h_A(100);

    goopax::goopax_device device = goopax::devices(goopax::env_CUDA)[0];

    goopax::kernel inc_goopax(device, [](goopax::resource<float>& A) {
        using namespace goopax;
        gpu_for_global(0, A.size(), [&](gpu_uint k) { A[k] += 1; });
    });

    for (unsigned int k = 0; k < h_A.size(); ++k)
    {
        h_A[k] = k;
    }
    cout << "starting with h_A=" << h_A << endl;

    cout << "Allocating cuda device memory." << endl;
    float* d_A = nullptr;
    check_cuda(hipMalloc((void**)&d_A, h_A.size() * sizeof(h_A[0])));

    cout << "Copying to cuda device memory." << endl;
    check_cuda(hipMemcpy(d_A, h_A.data(), h_A.size() * sizeof(float), hipMemcpyHostToDevice));

    cout << "Sharing memory region with goopax" << endl;
    goopax::buffer<float> A_goopax = goopax::buffer<float>::create_from_cuda(device, d_A, h_A.size());

    cout << "Calling cuda kernel" << endl;
    int threadsPerBlock = 256;
    int blocksPerGrid = 16;
    inc_cuda<<<blocksPerGrid, threadsPerBlock>>>(d_A, h_A.size());
    check_cuda(hipGetLastError());

    cout << "Calling goopax kernel" << endl;
    inc_goopax(A_goopax);

    cout << "Copying back to host memory." << endl;
    A_goopax.copy_to_host(h_A.data());

    cout << "now: h_A=" << h_A << endl;

    cout << "Checking result." << endl;
    for (unsigned int k = 0; k < h_A.size(); ++k)
    {
        assert(h_A[k] == k + 2);
    }

    cout << "Freeing cuda memory" << endl;
    check_cuda(hipFree(d_A));
}
